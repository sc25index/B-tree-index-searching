#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <vector>
#include <hipcub/hipcub.hpp>

#include "utils.h"
#include "grouping2d.h"


#define BLOCK_SIZE 768

#define BIT_PASS 3 // number of bit each time

#define BUCK_SIZE 256
#define BIT_MASK 0xF
#define CUDA_MAX_BLOCK 65535


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true);


__host__ __device__ inline int bucketIdentify2d(ulli input, ulli input2, int nPass)  {

    int maxPass1 = (int)((sizeof(input)*8)/BIT_PASS) + ((((sizeof(input)*8)%BIT_PASS)>0)?1:0);
    int maxPass2 = (int)((sizeof(input2)*8)/BIT_PASS) + ((((sizeof(input2)*8)%BIT_PASS)>0)?1:0);
    int num1 = (input>>(BIT_PASS*(maxPass1-1-nPass))) & BIT_MASK;
    int num2 = (input2>>(BIT_PASS*(maxPass2-1-nPass))) & BIT_MASK;
    return num1*num2;
}


__global__
void kernelRadixPass2d( const ulli *dev_input_, const ulli *dev_input2_,
                        ulli *dev_Hist, int nPass, ulli inputSize) {
    __shared__ int SHist[BUCK_SIZE];
    const ulli2* __restrict dev_input = (ulli2*)dev_input_;
    const ulli2* __restrict dev_input2 = (ulli2*)dev_input2_;
    ulli h_pos = threadIdx.x;
    while(h_pos < BUCK_SIZE) {
        SHist[h_pos]=0;
        h_pos += blockDim.x;
    }
    __syncthreads();

    h_pos = threadIdx.x + blockIdx.x * blockDim.x;
#if __CUDA_ARCH__  >= 750
    int twid = threadIdx.x%32;
#endif
    while( h_pos < inputSize/2 ) {
        ulli2 currentPtr = dev_input[h_pos];
        ulli2 currentPtr2 = dev_input2[h_pos];
#if __CUDA_ARCH__  >= 750
        if(((h_pos/32)*32)+32  < inputSize/2) {
            int bucketAns = bucketIdentify2d(currentPtr.a, currentPtr2.a, nPass);
            int bitmap = __match_any_sync(0xffffffff,bucketAns);
            if(__ffs(bitmap)-1 == twid) {
                atomicAdd(&SHist[bucketAns],__popc(bitmap));
            }
            bucketAns = bucketIdentify2d(currentPtr.b, currentPtr2.b,nPass);
            bitmap = __match_any_sync(0xffffffff,bucketAns);

            if(__ffs(bitmap)-1 == twid) {
                atomicAdd(&SHist[bucketAns],__popc(bitmap));
            }
        }
        else {
            int bucketAns = bucketIdentify2d(currentPtr.a, currentPtr2.a, nPass);
            atomicAdd(&SHist[bucketAns],1);
            bucketAns = bucketIdentify2d(currentPtr.b, currentPtr2.b,nPass);
            atomicAdd(&SHist[bucketAns],1);
        }
#else
        int bucketAns = bucketIdentify2d(currentPtr.a, currentPtr2.a, nPass);
        atomicAdd(&SHist[bucketAns],1);
        bucketAns = bucketIdentify2d(currentPtr.b, currentPtr2.b,nPass);
        atomicAdd(&SHist[bucketAns],1);
#endif

        h_pos += gridDim.x * blockDim.x;
    }

    {
        if (threadIdx.x == 0 && blockIdx.x == 0) {
            int remainder = inputSize%2;
            if(remainder != 0) {
                while(remainder) {
                    int idx = inputSize - remainder--;
                    int bucketAns = bucketIdentify2d(dev_input_[idx],dev_input2_[idx],nPass);
                    atomicAdd(&SHist[bucketAns],1);
                }
            }
        }
    }

    __syncthreads();
    h_pos = threadIdx.x;
    while( h_pos < BUCK_SIZE ) {
        atomicAdd(&dev_Hist[h_pos],SHist[h_pos]);
        h_pos += blockDim.x;
    }
}


__global__
void kernelRadixPassRelocate2d( const ulli *dev_input_, const ulli *dev_input2_, ulli *dev_output,
                                ulli *dev_output2, ulli* dev_histogram, ulli *dev_Prefix, int nPass,
                                ulli inputSize) {

    __shared__ ulli SHLocate[BUCK_SIZE];
    __shared__ int SHist[BUCK_SIZE];
    __shared__ ulli SBuffer[BLOCK_SIZE];
    __shared__ ulli SBuffer2[BLOCK_SIZE];
    __shared__ int SInLoopPrefix[BUCK_SIZE];
    __shared__ int SInLoopBKPrefix[BUCK_SIZE];

    const ulli* __restrict dev_input = (ulli*)dev_input_;
    const ulli* __restrict dev_input2 = (ulli*)dev_input2_;
    ulli h_pos = threadIdx.x;
    while(h_pos < BUCK_SIZE) {
        SHist[h_pos]=0;
        h_pos += blockDim.x;
    }

    __syncthreads();
    h_pos = threadIdx.x + blockIdx.x * blockDim.x;
#if __CUDA_ARCH__  >= 750
    int twid = threadIdx.x%32;
#endif
    while( h_pos < inputSize ) {

        ulli currentPtr = dev_input[h_pos];
        ulli currentPtr2 = dev_input2[h_pos];
#if __CUDA_ARCH__  >= 750
        if(((h_pos/32)*32)+32  < inputSize) {
            int bucketAns = bucketIdentify2d(currentPtr, currentPtr2, nPass);
            int bitmap = __match_any_sync(0xffffffff,bucketAns);
            if(__ffs(bitmap)-1 == twid) {
                atomicAdd(&SHist[bucketAns],__popc(bitmap));
            }
        }
        else {
            int bucketAns = bucketIdentify2d(currentPtr, currentPtr2, nPass);
            atomicAdd(&SHist[bucketAns],1);
        }
#else
        int bucketAns = bucketIdentify2d(currentPtr, currentPtr2, nPass);
        atomicAdd(&SHist[bucketAns],1);
#endif
        h_pos += gridDim.x * blockDim.x;
    }

    __syncthreads();

    h_pos = threadIdx.x;
    while(h_pos < BUCK_SIZE) {
        SHLocate[h_pos] = atomicAdd(&dev_Prefix[h_pos],SHist[h_pos]);
        h_pos += blockDim.x;
    }
    __syncthreads();

    // reuse histogram for int-loop histogram
    // assume threadid.x always greater than BUCK_SIZE !!!
    // Fixme enable vector 2
    // Fixme when the inputSize isn't diviable by blocksize (on the 4 round)
    h_pos = threadIdx.x + blockIdx.x * blockDim.x;
    while( h_pos < inputSize ) {


        //reset inloop histogram
        ulli h_pos2 = threadIdx.x;
        while(h_pos2 < BUCK_SIZE) {
            SHist[h_pos2]=0;
            if( inputSize%BLOCK_SIZE != 0 && (h_pos-threadIdx.x+BLOCK_SIZE) > inputSize)
                h_pos2 += (inputSize%BLOCK_SIZE);
            else
                h_pos2 += blockDim.x;
        }
        SBuffer[threadIdx.x] = 0;

        __syncthreads();

        ulli currentPtr = dev_input[h_pos];
        ulli currentPtr2 = dev_input2[h_pos];
        int bucketAns = bucketIdentify2d(currentPtr,currentPtr2,nPass);
        atomicAdd(&SHist[bucketAns],1);

        __syncthreads();

        //calcuate in loop prefix
        if(threadIdx.x == 0) {
            SInLoopPrefix[0] = 0;
            for(int i=1;i<BUCK_SIZE;i++) {
                SInLoopPrefix[i] = SInLoopPrefix[i-1] + SHist[i-1];
            }
        }

        __syncthreads();

        h_pos2 = threadIdx.x;
        while(h_pos2 < BUCK_SIZE) {
            SInLoopBKPrefix[h_pos2] = SInLoopPrefix[h_pos2];
            if( inputSize%BLOCK_SIZE != 0 && (h_pos-threadIdx.x+BLOCK_SIZE) > inputSize)
                h_pos2 += (inputSize%BLOCK_SIZE);
            else
                h_pos2 += blockDim.x;
        }

        __syncthreads();

        ulli offset = atomicAdd(&SInLoopPrefix[bucketAns],1);
        SBuffer[offset] = currentPtr;
        SBuffer2[offset] = currentPtr2;

        __syncthreads();

        {
            currentPtr = SBuffer[threadIdx.x];
            currentPtr2 = SBuffer2[threadIdx.x];
            bucketAns = bucketIdentify2d(currentPtr,currentPtr2,nPass);
            offset = SHLocate[bucketAns]+ threadIdx.x - SInLoopBKPrefix[bucketAns];
            dev_output[offset] = currentPtr;
            dev_output2[offset] = currentPtr2;
        }

        __syncthreads();

        h_pos2 = threadIdx.x;
        while(h_pos2 < BUCK_SIZE) {
            SHLocate[h_pos2] += SHist[h_pos2];
            if( inputSize%BLOCK_SIZE != 0 && (h_pos-threadIdx.x+BLOCK_SIZE) > inputSize)
                h_pos2 += (inputSize%BLOCK_SIZE);
            else
                h_pos2 += blockDim.x;
        }

        h_pos += (gridDim.x * blockDim.x);
    }

}

__global__
void kernelRadixPassSingleKernel2d( const ulli *dev_input_, const ulli *dev_input2_, ulli *dev_prefix,
                                    ulli *dev_group_cout, int nPass, ulli *dev_output, ulli *dev_output2,
                                    ulli group_num, ulli *dev_groupNum_out, ulli Local_Group_Size) {
    __shared__ int SHist[BUCK_SIZE];
    __shared__ int SPrefix[BUCK_SIZE];

    ulli currentGroup = blockIdx.x;
    ulli h_pos;

    while(currentGroup < group_num){
        const ulli* __restrict dev_input = &dev_input_[dev_prefix[currentGroup]];
        const ulli* __restrict dev_input2 = &dev_input2_[dev_prefix[currentGroup]];
        if(dev_group_cout[currentGroup] > Local_Group_Size) {
            h_pos = threadIdx.x;
            while(h_pos < BUCK_SIZE) {
                SHist[h_pos]=0;
                SPrefix[h_pos]=0;
                h_pos += blockDim.x;
            }
            __syncthreads();

            h_pos = threadIdx.x;
#if __CUDA_ARCH__  >= 750
            int twid = threadIdx.x%32;
#endif
            // calcuate histogram
            while( h_pos < dev_group_cout[currentGroup] ) {

                ulli currentPtr = dev_input[h_pos];
                ulli currentPtr2 = dev_input2[h_pos];
#if __CUDA_ARCH__  >= 750
                if(((h_pos/32)*32)+32  < dev_group_cout[currentGroup]) {
                    int bucketAns = bucketIdentify2d(currentPtr, currentPtr2, nPass);
                    int bitmap = __match_any_sync(0xffffffff,bucketAns);
                    if(__ffs(bitmap)-1 == twid) {
                        atomicAdd(&SHist[bucketAns],__popc(bitmap));
                    }
                }
                else {
                    int bucketAns = bucketIdentify2d(currentPtr, currentPtr2, nPass);
                    atomicAdd(&SHist[bucketAns],1);
                }
#else
                int bucketAns = bucketIdentify2d(currentPtr, currentPtr2, nPass);
                atomicAdd(&SHist[bucketAns],1);
#endif
                h_pos += blockDim.x;
            }

            __syncthreads();

            h_pos = threadIdx.x;
            ulli *currentGroupList = &dev_groupNum_out[BUCK_SIZE*currentGroup];
            while(h_pos < BUCK_SIZE) {
                currentGroupList[h_pos] = SHist[h_pos];
                h_pos += blockDim.x;
            }

            // calcuate prefix sum
            if( threadIdx.x == 0 ) {
                SPrefix[0]=0;
                for(int i=1;i<BUCK_SIZE;i++){
                    SPrefix[i]=SPrefix[i-1] + SHist[i-1];
                }
            }

            __syncthreads();

            h_pos = threadIdx.x;
            while( h_pos < dev_group_cout[currentGroup] ) {

                ulli currentPtr = dev_input[h_pos];
                ulli currentPtr2 = dev_input2[h_pos];
                int bucketAns = bucketIdentify2d(currentPtr,currentPtr2,nPass);
                int offset = atomicAdd(&SPrefix[bucketAns],1);
                dev_output[dev_prefix[currentGroup]+offset] = currentPtr;
                dev_output2[dev_prefix[currentGroup]+offset] = currentPtr2;
                h_pos += (blockDim.x);
            }

            __syncthreads();
        }
        else {
            if( threadIdx.x == 0 ) {
                ulli *currentGroupList = &dev_groupNum_out[BUCK_SIZE*currentGroup];
                currentGroupList[0] = dev_group_cout[currentGroup];
            }
            h_pos = threadIdx.x;
            while( h_pos < dev_group_cout[currentGroup] ) {
                dev_output[dev_prefix[currentGroup]+h_pos] = dev_input[h_pos];
                dev_output2[dev_prefix[currentGroup]+h_pos] = dev_input2[h_pos];
                h_pos += (blockDim.x);
            }

            __syncthreads();
        }
        currentGroup += gridDim.x;
    }

}


// groupNum is number of groupt
// groupList is size of each group
// groupPrefix is prefix of each group in dev_input
void radixGroupGPU2d(ulli *dev_input, ulli *dev_input2, ulli *dev_output, ulli *dev_output2,
                    std::vector<ulli> &groupList, std::vector<ulli> &groupPrefix,
                    int pass, ulli Local_Group_Size) {

    std::vector<ulli> currentList;
    std::vector<ulli> currentPrefix;

    for(int i=0;i<groupList.size();i++) {
        if(groupList[i] == 0) {
            continue;
        }
        else if(groupList[i] < Local_Group_Size) {
            currentList.push_back(groupList[i]);
            currentPrefix.push_back(groupPrefix[i]);
            hipMemcpyAsync( dev_output, dev_input, sizeof(ulli)*groupList[i], hipMemcpyDeviceToDevice);
            hipMemcpyAsync( dev_output2, dev_input2, sizeof(ulli)*groupList[i], hipMemcpyDeviceToDevice);
            continue;
        }

        ulli *dev_histogram;
        ulli *devPrefix;
        ulli *host_Prefix = new ulli[BUCK_SIZE];
        ulli *host_histogram = new ulli[BUCK_SIZE];

        hipMalloc( (void**) &(dev_histogram), sizeof(ulli)*BUCK_SIZE);
        hipMalloc( (void**) &devPrefix, sizeof(ulli)*BUCK_SIZE);

        hipMemset( devPrefix, 0, sizeof(ulli)*BUCK_SIZE );
        hipMemset( dev_histogram, 0, sizeof(ulli)*BUCK_SIZE );

        //std::cout<<"Number of input "<<groupList[i]<<std::endl;

        ulli m_blocks = (groupList[i] / (BLOCK_SIZE))>CUDA_MAX_BLOCK ? CUDA_MAX_BLOCK:groupList[i] /(BLOCK_SIZE);
        dim3 grid(m_blocks, 1, 1);
        dim3 threads(BLOCK_SIZE, 1, 1);

        //std::cout<<"Number of Block "<<m_blocks<<std::endl;
        //std::cout<<"Number of pass "<<pass<<std::endl;


        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipStreamSynchronize(0) );

        kernelRadixPass2d<<<grid,threads>>>(&dev_input[groupPrefix[i]],&dev_input2[groupPrefix[i]],
                                            dev_histogram,pass, groupList[i]);


        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipStreamSynchronize(0) );


        hipMemcpyAsync( host_histogram, dev_histogram, sizeof(ulli)*BUCK_SIZE, hipMemcpyDeviceToHost);

        dim3 grid2(1, 1, 1);
        dim3 threads2(512, 1, 1);
        kernelPrefixSum<<<grid2,threads2>>>(dev_histogram,devPrefix);


        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipStreamSynchronize(0) );

        int validBucket=0;
        for(int j=0;j<BUCK_SIZE;j++) {
            if(host_histogram[j] != 0) {
                validBucket++;
            }
        }

        if(validBucket == 1 && groupList.size() == 1) {
            currentList.push_back(groupList[i]);
            currentPrefix.push_back(groupPrefix[i]);
        }
        else {
            //note must copy prefix before call relocate!!!
            hipMemcpy( host_Prefix, devPrefix, sizeof(ulli)*BUCK_SIZE, hipMemcpyDeviceToHost);
            kernelRadixPassRelocate2d<<<grid,threads>>>(&dev_input[groupPrefix[i]],&dev_input2[groupPrefix[i]],
                                                        &dev_output[groupPrefix[i]],&dev_output2[groupPrefix[i]],
                                                        dev_histogram, devPrefix, pass, groupList[i]);

            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipStreamSynchronize(0) );


            for(int j=0;j<BUCK_SIZE;j++) {
                if(host_histogram[j] != 0 ) {
                    ulli k=0;
                    ulli currentCount = host_histogram[j];
                    while(currentCount < Local_Group_Size && (j+k+1) < BUCK_SIZE) {
                        if(currentCount+host_histogram[j+k+1]<Local_Group_Size) {
                            currentCount = currentCount + host_histogram[j+k+1];
                            k++;
                        }
                        else{
                            break;
                        }
                    }
                    currentList.push_back(currentCount);
                    currentPrefix.push_back(groupPrefix[i]+host_Prefix[j]);
                    j += k;
                }
            }
        }

        hipFree(dev_histogram);
        hipFree(devPrefix);

        delete host_Prefix;
        delete host_histogram;
    }

    groupList = currentList;
    groupPrefix = currentPrefix;
}


void radixGroupSingleKernelGPU2d(ulli *dev_input, ulli *dev_input2, ulli *dev_output, ulli *dev_output2,
                                 std::vector<ulli> &groupList, std::vector<ulli> &groupPrefix,
                                 int pass, ulli Local_Group_Size) {

    std::vector<ulli> currentList;
    std::vector<ulli> currentPrefix;

    ulli *dev_group_cout, *host_group_count;
    ulli *dev_prefix, *host_prefix;
    ulli *dev_groupOutList, *host_groupOutList;

    host_group_count = new ulli[groupList.size()];
    host_prefix = new ulli[groupList.size()];
    host_groupOutList = new ulli[BUCK_SIZE*groupList.size()];
    //std::cout<<"Allocation Complete!!"<<std::endl;

    ulli tmpcheck = 0;
    for(ulli i=0;i<groupList.size();i++) {
        host_group_count[i] = groupList[i];
        host_prefix[i] = groupPrefix[i];
        tmpcheck += groupList[i];
    }
    //std::cout<<"tmpcheck = "<<tmpcheck<<std::endl;
    hipMalloc( (void**) &(dev_group_cout), sizeof(ulli)*groupList.size());
    hipMalloc( (void**) &(dev_prefix), sizeof(ulli)*groupList.size());
    hipMalloc( (void**) &(dev_groupOutList), sizeof(ulli)*BUCK_SIZE*groupList.size());
    //std::cout<<"CUDA Allocation Complete!!"<<std::endl;

    hipMemcpy( dev_group_cout, host_group_count, sizeof(ulli)*groupList.size(), hipMemcpyHostToDevice);
    hipMemcpy( dev_prefix, host_prefix, sizeof(ulli)*groupList.size(), hipMemcpyHostToDevice);
    hipMemset( dev_groupOutList, 0, sizeof(ulli)*BUCK_SIZE*groupList.size() );

    gpuErrchk( hipPeekAtLastError() );

    ulli m_blocks = groupList.size()>CUDA_MAX_BLOCK ? CUDA_MAX_BLOCK:groupList.size();
    dim3 grid(m_blocks, 1, 1);
    dim3 threads(BLOCK_SIZE, 1, 1);
    //std::cout<<"Number of Group: "<<groupList.size()<<std::endl;
    //std::cout<<"Block size: "<<m_blocks<<std::endl;
    kernelRadixPassSingleKernel2d<<<grid,threads>>>( dev_input, dev_input2, dev_prefix, dev_group_cout,
                                                     pass, dev_output, dev_output2, groupList.size(),dev_groupOutList,
                                                     Local_Group_Size);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipStreamSynchronize(0) );

    hipMemcpy( host_groupOutList, dev_groupOutList, sizeof(ulli)*BUCK_SIZE*groupList.size(), hipMemcpyDeviceToHost);

    ulli prefixTmp=0;
    for(ulli i = 0;i<BUCK_SIZE*groupList.size();i++) {
        if(host_groupOutList[i] != 0) {
            ulli j=0;
            ulli currentCount = host_groupOutList[i];
            while(currentCount < Local_Group_Size && (i+j+1) < BUCK_SIZE*groupList.size()) {
                if(currentCount+host_groupOutList[i+j+1]<Local_Group_Size) {
                    currentCount = currentCount + host_groupOutList[i+j+1];
                    j++;
                }
                else{
                    break;
                }
            }
            currentList.push_back(currentCount);
            currentPrefix.push_back(prefixTmp);
            prefixTmp += currentCount;
            i = i+j;
        }
    }

    hipFree(dev_group_cout);
    hipFree(dev_groupOutList);
    hipFree(dev_prefix);
    delete host_group_count;
    delete host_prefix;
    delete host_groupOutList;

    groupList = currentList;
    groupPrefix = currentPrefix;
}


void genGrouping2d(ulli *input, ulli* input2, ulli inputSize, std::vector<ulli> &groupList,
                    ulli *&output, ulli* &output2, std::vector<ulli> &groupPrefix,
                    ulli Local_Group_Size) {

    ulli *host_output = new ulli[inputSize];
    ulli *host_output2 = new ulli[inputSize];
    output = host_output;
    output2 = host_output2;
    groupList.push_back(inputSize);
    groupPrefix.push_back(0);

    // init cuda
    ulli *dev_input;
    ulli *dev_input2;
    ulli *dev_output;
    ulli *dev_output2;

    hipEvent_t m_start, m_stop;
    float m_time;

    hipSetDevice(0);

    hipMalloc( (void**) &(dev_input), sizeof(ulli)*inputSize);
    hipMalloc( (void**) &(dev_input2), sizeof(ulli)*inputSize);
    hipMalloc( (void**) &(dev_output), sizeof(ulli)*inputSize);
    hipMalloc( (void**) &(dev_output2), sizeof(ulli)*inputSize);
    hipMemcpy( dev_input, input, sizeof(ulli)*inputSize, hipMemcpyHostToDevice);
    hipMemcpy( dev_input2, input2, sizeof(ulli)*inputSize, hipMemcpyHostToDevice);
    hipMemset( dev_output, 0, sizeof(ulli)*inputSize );
    hipMemset( dev_output2, 0, sizeof(ulli)*inputSize );


    hipEventCreate( &m_start );
    hipEventCreate( &m_stop );
    hipEventRecord( m_start, 0 );

    ulli *maxValueA;
    ulli *maxValueB;

    {
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipMallocManaged( (void**) &(maxValueA), sizeof(ulli));
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dev_input, maxValueA, inputSize);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run max-reduction
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dev_input, maxValueA, inputSize);

        hipDeviceSynchronize();

        hipFree(d_temp_storage);

    }

    {
        void *d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipMallocManaged( (void**) &(maxValueB), sizeof(ulli));
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dev_input2, maxValueB, inputSize);
        // Allocate temporary storage
        hipMalloc(&d_temp_storage, temp_storage_bytes);
        // Run max-reduction
        hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, dev_input2, maxValueB, inputSize);

        hipDeviceSynchronize();

        hipFree(d_temp_storage);
    }

    int MaxBitA = 0;
    int MaxBitB = 0;
    while(*maxValueA) {
        *maxValueA = (*maxValueA)>>1;
        MaxBitA++;
    }
    while(*maxValueB) {
        *maxValueB = (*maxValueB)>>1;
        MaxBitB++;
    }

    int StartPassA = 16-ceil((float)MaxBitA/(float)BIT_PASS);
    int StartPassB = 16-ceil((float)MaxBitB/(float)BIT_PASS);

    int pass=0;
    if(StartPassA > StartPassB)
        pass = StartPassA;
    else
        pass = StartPassB;

    for(;pass<16;pass++){
        if(groupList.size() != 1){
            ulli *tmpDev = dev_input;
            dev_input = dev_output;
            dev_output = tmpDev;

            tmpDev = dev_input2;
            dev_input2 = dev_output2;
            dev_output2 = tmpDev;
        }

        if( groupList.size() < 10 )
            radixGroupGPU2d(dev_input, dev_input2, dev_output, dev_output2, groupList, groupPrefix, pass, Local_Group_Size);
        else
            radixGroupSingleKernelGPU2d(dev_input, dev_input2, dev_output, dev_output2, groupList, groupPrefix, pass, Local_Group_Size);

        double averageSize = 0;
        ulli MinSize=groupList[0];
        ulli MaxSize=groupList[0];
        for(int i=0;i<groupList.size();i++){
            averageSize += groupList[i];
            if(MinSize > groupList[i])
                MinSize = groupList[i];
            if(MaxSize < groupList[i])
                MaxSize = groupList[i];
        }

        averageSize = averageSize/groupList.size();
#if 1

        double standardDeviation = 0;
        for(int i=0;i<groupList.size();i++)
            standardDeviation += pow(groupList[i] - averageSize, 2);
        standardDeviation = sqrt(standardDeviation / 10);

        std::cout<<"Number of passed "<<pass<<" Total Group: "<<groupList.size()<<" averageSize: "<<averageSize<<std::endl;
        std::cout<<"StandardDeviation: "<<standardDeviation<<" Max: "<<MaxSize<<" Min: "<<MinSize <<std::endl;
#endif


        if(MaxSize <= Local_Group_Size) {
            break;
        }
    }

    hipDeviceSynchronize();


    hipDeviceSynchronize();
    hipEventRecord( m_stop, 0 );
    hipEventSynchronize( m_stop );
    hipEventElapsedTime( &m_time, m_start, m_stop);
    hipEventDestroy( m_start);
    hipEventDestroy( m_stop);

    hipMemcpy( host_output, dev_output, sizeof(ulli)*inputSize, hipMemcpyDeviceToHost);
    hipMemcpy( host_output2, dev_output2, sizeof(ulli)*inputSize, hipMemcpyDeviceToHost);


    hipFree(dev_input);
    hipFree(dev_input2);
    hipFree(dev_output);
    hipFree(dev_output2);

    //for(int i=0;i<groupList[0];i++){
    //    std::cout<<"Debug:"<<host_output[i]<<"\n";
    //}

#if 0
    for(int i=0;i<groupList[0];i++){
        std::cout<<"Debug:"<<host_output[i]<<"\n";
    }

    for(int i=0;i<groupList[1];i++){
        std::cout<<"Debug2:"<<host_output[groupList[0]+i]<<"\n";
    }
#endif

#if 0
    std::cout<<"Testing Result"<<std::endl;
    for(ulli i =0;i<groupList.size()-1;i++) {
        #pragma omp parallel for
        for(ulli j=groupPrefix[i];j<groupPrefix[i+1];j++) {
            for(ulli k=groupPrefix[i+1];k<inputSize;k++) {
                if(host_output[j]>host_output[k]) {
                    std::cout<<"Negitive: ";
                    std::cout<<host_output[j]<<" Belong to group "<<i << " at "<< j <<" greater than "<<host_output[k] << "At "<<k<<" End at:" <<groupPrefix[i+1]<<std::endl;
                    break;
                }
            }
        }
        if(i%100 ==0)
            std::cout<<"Group Pass:"<<i<<std::endl;
    }
#endif
    //std::cout<<"Number of group = "<<groupList.size()<<std::endl;
    //ulli checkNum=0;
    //for(int i=0;i<groupList.size();i++){
        //if(test[i]<LOCAL_BLOCK_SIZE/2)
        //    std::cout<<"Block Number "<<i<<" less than "<< LOCAL_BLOCK_SIZE/2 <<" which is "<< test[i] <<std::endl;
        //checkNum+=groupList[i];
    //}
    //std::cout<<"Total Output:"<<checkNum<<std::endl;

    //std::cout<<"Running time:"<<m_time/1000<<"Sec."<<std::endl;
    std::cout<<"grouping2d = "<<m_time<<std::endl;

    return;
}




inline void gpuAssert(hipError_t code, const char *file, int line, bool abort)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
